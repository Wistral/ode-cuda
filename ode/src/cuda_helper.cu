#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <ode/common.h>
#include <ode/cuda_helper.h>

ODE_API void cuda_checkError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err));
		exit(EXIT_FAILURE);
	}                         
}

ODE_API void cuda_testMemcpy()
{
	float *a_h, *b_h;	// pointers to host memory
	float *a_d, *b_d;	// pointers to device memory
	int N = 14;
	int i;
	// allocate arrays on host
	a_h = (float*) malloc(sizeof(float)*N);
	b_h = (float*) malloc(sizeof(float)*N);
	// allocate arrays on device
	hipMalloc((void**) &a_d, sizeof(float)*N);
	hipMalloc((void**) &b_d, sizeof(float)*N);
	// initialize host data
	for (i=0; i<N; i++) {
		a_h[i] = 10.f+i;
		b_h[i] = 0.f;
	}
	// send data from host to device: a_h to a_d
	hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
	// copy data within device: a_d to b_d
	hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
	// retrieve data from device: b_d to b_h
	hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
	// check result
	for (i=0; i<N; i++)
		assert(a_h[i] == b_h[i]);
	// cleanup
	free(a_h); free(b_h);
	hipFree(a_d); hipFree(b_d);
}

ODE_API dReal *cuda_copyToDevice(dReal *a, int n)
{
	dReal *dev_a;
	hipMalloc((void**) &dev_a, sizeof(dReal)*n);
	cuda_checkError("malloc");
	hipMemcpy(dev_a, a, sizeof(dReal)*n, hipMemcpyHostToDevice);
	cuda_checkError("memcpy h to d");
	return dev_a;
}

ODE_API dReal *cuda_copyFromDevice(dReal *dev_a, dReal *a, int n)
{
	hipMemcpy(a, dev_a, sizeof(dReal)*n, hipMemcpyDeviceToHost);
	cuda_checkError("memcpy d to h");
	return a;
}

ODE_API void cuda_freeFromDevice(dReal *dev_a)
{
	hipFree(dev_a);
}

ODE_API dReal *cuda_makeOnDevice(int n)
{
	dReal *dev_a;
	hipMalloc((void**) &dev_a, sizeof(dReal)*n);
	cuda_checkError("malloc");
	return dev_a;
}

