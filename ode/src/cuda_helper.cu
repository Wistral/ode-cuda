#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <ode/common.h>
#include "objects.h"
#include <ode/cuda_helper.h>

ODE_API void cuda_checkError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err));
		exit(EXIT_FAILURE);
	}                         
}

ODE_API void cuda_testMemcpy()
{
	float *a_h, *b_h;	// pointers to host memory
	float *a_d, *b_d;	// pointers to device memory
	int N = 14;
	int i;
	// allocate arrays on host
	a_h = (float*) malloc(sizeof(float)*N);
	b_h = (float*) malloc(sizeof(float)*N);
	// allocate arrays on device
	hipMalloc((void**) &a_d, sizeof(float)*N);
	hipMalloc((void**) &b_d, sizeof(float)*N);
	// initialize host data
	for (i=0; i<N; i++) {
		a_h[i] = 10.f+i;
		b_h[i] = 0.f;
	}
	// send data from host to device: a_h to a_d
	hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
	// copy data within device: a_d to b_d
	hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
	// retrieve data from device: b_d to b_h
	hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
	// check result
	for (i=0; i<N; i++)
		assert(a_h[i] == b_h[i]);
	// cleanup
	free(a_h); free(b_h);
	hipFree(a_d); hipFree(b_d);
}

ODE_API dReal *cuda_copyToDevice(dReal *a, int n)
{
	dReal *dev_a;
	hipMalloc((void**) &dev_a, sizeof(dReal)*n);
	cuda_checkError("malloc");
	hipMemcpy(dev_a, a, sizeof(dReal)*n, hipMemcpyHostToDevice);
	cuda_checkError("memcpy h to d");
	return dev_a;
}

ODE_API dReal *cuda_copyFromDevice(dReal *dev_a, dReal *a, int n)
{
	hipMemcpy(a, dev_a, sizeof(dReal)*n, hipMemcpyDeviceToHost);
	cuda_checkError("memcpy d to h");
	return a;
}

ODE_API void cuda_freeFromDevice(dReal *dev_a)
{
	hipFree(dev_a);
}

ODE_API dReal *cuda_makeOnDevice(int n)
{
	dReal *dev_a;
	hipMalloc((void**) &dev_a, sizeof(dReal)*n);
	cuda_checkError("malloc");
	return dev_a;
}

ODE_API dxBody *cuda_copyBodiesToDevice(dxBody *cuda_body, dxBody **body, int NUM)
{
	int i;
	for (i=0;i<NUM;i++) {
		hipMemcpy(cuda_body+i, body[i], sizeof(dxBody), hipMemcpyHostToDevice);
		printf("%f\n", body[i]->posr.pos[0]);
	}
	cuda_checkError("memcpy h to d");
	return cuda_body;
}

ODE_API dxBody **cuda_copyBodiesFromDevice(dxBody **body, dxBody *cuda_body, int NUM)
{
	int i;
	for (i=0;i<NUM;i++) {
/*		free(body[i]);
		dxBody *b = (dxBody *) malloc(sizeof(dxBody));
		hipMemcpy(b, cuda_body+i, sizeof(dxBody), hipMemcpyDeviceToHost);
		body[i] = b;*/
		hipMemcpy(body[i], cuda_body+i, sizeof(dxBody), hipMemcpyDeviceToHost);
	}
	cuda_checkError("memcpy d to h");
	return body;
}

ODE_API dxBody *cuda_initBodiesOnDevice(int NUM)
{
	printf("%i\n", sizeof(dxBody));
	dxBody *cuda_body;
	hipMalloc((void**) &cuda_body, sizeof(dxBody)*NUM);
	cuda_checkError("malloc");
	return cuda_body;
}

ODE_API void cuda_free(dxBody *ptr)
{
	hipFree(ptr);
}

