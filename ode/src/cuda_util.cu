#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <ode/common.h>
#include <ode/cuda_helper.h>
#include <ode/cuda_matrix.h>
#include "util.h"
#include "cuda_util.h"

#define BLOCKSIZE 4

struct cuda_Island {
	dxBody **body;
	dxJoint **joint;
	int nb;
	int nj;
}

sizeof(cuda_Island) * world->nb

template <int BLOCK_SIZE> __global__ void process_islands()
{
  dxBody *b,*bb,**body;
  dxJoint *j,**joint;

  // nothing to do if no bodies
  if (world->nb <= 0) return;

  // handle auto-disabling of bodies
  dInternalHandleAutoDisabling (world,stepsize);

  // make arrays for body and joint lists (for a single island) to go into
  body = (dxBody**) ALLOCA (world->nb * sizeof(dxBody*));
  joint = (dxJoint**) ALLOCA (world->nj * sizeof(dxJoint*));
  int bcount = 0;	// number of bodies in `body'
  int jcount = 0;	// number of joints in `joint'

  // set all body/joint tags to 0
  for (b=world->firstbody; b; b=(dxBody*)b->next) b->tag = 0;
  for (j=world->firstjoint; j; j=(dxJoint*)j->next) j->tag = 0;

  // allocate a stack of unvisited bodies in the island. the maximum size of
  // the stack can be the lesser of the number of bodies or joints, because
  // new bodies are only ever added to the stack by going through untagged
  // joints. all the bodies in the stack must be tagged!
  int stackalloc = (world->nj < world->nb) ? world->nj : world->nb;
  dxBody **stack = (dxBody**) ALLOCA (stackalloc * sizeof(dxBody*));

  islands = ALLOCA (world->nb * sizeof(cuda_Island));

  for (bb=world->firstbody; bb; bb=(dxBody*)bb->next) {
    // get bb = the next enabled, untagged body, and tag it
    if (bb->tag || (bb->flags & dxBodyDisabled)) continue;
    bb->tag = 1;

    // tag all bodies and joints starting from bb.
    int stacksize = 0;
    b = bb;
    body[0] = bb;
    bcount = 1;
    jcount = 0;
    goto quickstart;
    while (stacksize > 0) {
      b = stack[--stacksize];	// pop body off stack
      body[bcount++] = b;	// put body on body list
      quickstart:

      // traverse and tag all body's joints, add untagged connected bodies
      // to stack
      for (dxJointNode *n=b->firstjoint; n; n=n->next) {
        if (!n->joint->tag && n->joint->isEnabled()) {
	  n->joint->tag = 1;
	  joint[jcount++] = n->joint;
	  if (n->body && !n->body->tag) {
	    n->body->tag = 1;
	    stack[stacksize++] = n->body;
	  }
	}
      }
      dIASSERT(stacksize <= world->nb);
      dIASSERT(stacksize <= world->nj);
    }

	islands[memcpy
	islands[island_count++] = 
  }

    // now do something with body and joint lists
	//stepper (world,island->body,island-nb,island->joint,island->nj,stepsize);
    stepper (world,body,bcount,joint,jcount,stepsize);

    // what we've just done may have altered the body/joint tag values.
    // we must make sure that these tags are nonzero.
    // also make sure all bodies are in the enabled state.
    int i;
    for (i=0; i<bcount; i++) {
      body[i]->tag = 1;
      body[i]->flags &= ~dxBodyDisabled;
    }
    for (i=0; i<jcount; i++) joint[i]->tag = 1;
  }
}

ODE_API void cuda_dxProcessIslands(dxWorld *world, dReal stepsize, dstepper_fn_t stepper)
{
	const int block_size = BLOCKSIZE;
	dim3 dimBlock(block_size, block_size);
	dim3 dimGrid(block_size, block_size);
	process_islands<block_size><<<dimGrid, dimBlock>>>();
}

