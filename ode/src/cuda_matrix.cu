#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <ode/common.h>
#include <ode/cuda_matrix.h>
#include "util.h"
#include "config.h"

void cuda_checkError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err));
		exit(EXIT_FAILURE);
	}                         
}

void cuda_testMemcpy()
{
	float *a_h, *b_h;	// pointers to host memory
	float *a_d, *b_d;	// pointers to device memory
	int N = 14;
	int i;
	// allocate arrays on host
	a_h = (float*) malloc(sizeof(float)*N);
	b_h = (float*) malloc(sizeof(float)*N);
	// allocate arrays on device
	hipMalloc((void**) &a_d, sizeof(float)*N);
	hipMalloc((void**) &b_d, sizeof(float)*N);
	// initialize host data
	for (i=0; i<N; i++) {
		a_h[i] = 10.f+i;
		b_h[i] = 0.f;
	}
	// send data from host to device: a_h to a_d
	hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
	// copy data within device: a_d to b_d
	hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
	// retrieve data from device: b_d to b_h
	hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
	// check result
	for (i=0; i<N; i++)
		assert(a_h[i] == b_h[i]);
	// cleanup
	free(a_h); free(b_h);
	hipFree(a_d); hipFree(b_d);
}

dReal *cuda_copyToDevice(dReal *a, int n)
{
	dReal *dev_a;
	hipMalloc((void**) &dev_a, sizeof(dReal)*n);
	cuda_checkError("malloc");
	hipMemcpy(dev_a, a, sizeof(dReal)*n, hipMemcpyHostToDevice);
	cuda_checkError("memcpy");
	return dev_a;
}

dReal *cuda_copyFromDevice(dReal *dev_a, dReal *a, int n)
{
	hipMemcpy(a, dev_a, sizeof(float)*n, hipMemcpyDeviceToHost);
	cuda_checkError("memcpy");
	return a;
}

void cuda_freeFromDevice(dReal *dev_a)
{
	hipFree(dev_a);
}

__global__ void setzero(dReal *a, int n)
{
	int tid = blockIdx.x;
	if(tid < n)
		a[tid] = 0;
}

__global__ void setvalue(dReal *a, int n, dReal value)
{
	int tid = blockIdx.x;
	if(tid < n)
		a[tid] = value;
}

void cuda_dSetZero2(dReal *a, int n)
{
	dReal *dev_a; 

	// allocate memory on GPU
	hipMalloc((void**) &dev_a, n*sizeof(dReal));
	cuda_checkError("malloc");

	//copy array from CPU to GPU (not necessary)
	hipMemcpy(dev_a, a, n*sizeof(dReal), hipMemcpyHostToDevice);
	cuda_checkError("memcpy");

	//fill array with 0 on the gpu
	setzero<<<n,1>>>(dev_a, n);

	//copy array of 0's 'a' from GPU to CPU
	hipMemcpy(a, dev_a, n*sizeof(dReal), hipMemcpyDeviceToHost);
	cuda_checkError("memcpy");

	hipFree(dev_a);		
}

void cuda_dSetZero(dReal *dev_a, int n)
{
	setzero<<<n,1>>>(dev_a, n);
}

void cuda_dSetValue(dReal *dev_a, int n, dReal value)
{
	setvalue<<<n,1>>>(dev_a, n, value);
}

void cuda_dMultiply0(dReal *dev_A, const dReal *dev_B, const dReal *dev_c, int p, int q, int r)
{
}

void cuda_dMultiply1(dReal *dev_A, const dReal *dev_B, const dReal *dev_c, int p, int q, int r)
{
}

void cuda_dMultiply2(dReal *dev_A, const dReal *dev_B, const dReal *dev_c, int p, int q, int r)
{
}
