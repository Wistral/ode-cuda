#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <ode/common.h>
#include <ode/cuda_matrix.h>
#include "util.h"
#include "config.h"

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err));
		exit(EXIT_FAILURE);
	}                         
}

void cuda_testMemcpy()
{
	float *a_h, *b_h;	// pointers to host memory
	float *a_d, *b_d;	// pointers to device memory
	int N = 14;
	int i;
	// allocate arrays on host
	a_h = (float*) malloc(sizeof(float)*N);
	b_h = (float*) malloc(sizeof(float)*N);
	// allocate arrays on device
	hipMalloc((void**) &a_d, sizeof(float)*N);
	hipMalloc((void**) &b_d, sizeof(float)*N);
	// initialize host data
	for (i=0; i<N; i++) {
		a_h[i] = 10.f+i;
		b_h[i] = 0.f;
	}
	// send data from host to device: a_h to a_d
	hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
	// copy data within device: a_d to b_d
	hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
	// retrieve data from device: b_d to b_h
	hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
	// check result
	for (i=0; i<N; i++)
		assert(a_h[i] == b_h[i]);
	// cleanup
	free(a_h); free(b_h);
	hipFree(a_d); hipFree(b_d);
}

__global__ void setzero(dReal *a, int n)
{
	int tid = blockIdx.x;
	if(tid < n)
		a[tid] = 0;
}

void cuda_dSetZero(dReal *a, int n)
{
	dReal *dev_a; 

	// allocate memory on GPU
	hipMalloc((void**) &dev_a, n*sizeof(dReal));
	checkCUDAError("malloc");

	//copy array from CPU to GPU (not necessary)
	hipMemcpy(dev_a, a, n*sizeof(dReal), hipMemcpyHostToDevice);
	checkCUDAError("memcpy");

	//fill array with 0 on the gpu
	setzero<<<n,1>>>(dev_a, n);

	//copy array of 0's 'a' from GPU to CPU
	hipMemcpy(a, dev_a, n*sizeof(dReal), hipMemcpyDeviceToHost);
	checkCUDAError("memcpy");

	hipFree(dev_a);		
}

