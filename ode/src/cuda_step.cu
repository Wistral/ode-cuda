#include "hip/hip_runtime.h"
// cuda_step.cu

//#include "objects.h"
//#include "joints/joint.h"
//#include <ode/odeconfig.h>
//#include "config.h"
//#include <ode/odemath.h>
//#include <ode/rotation.h>
//#include <ode/timer.h>
//#include <ode/error.h>
//#include <ode/matrix.h>
//#include "lcp.h"
#include "util.h"

#include <ode/cuda_step.h>
#include <hip/hip_runtime.h>
#include <ode/cuda_helper.h>
// #include <ode/cuda_demo_helper.h>
#include <ode/cuda_matrix.h>
// #include <ode/cuPrintf.cuh>

#define BLOCKSIZE 16

__device__ void printMatrixBase(const char *name, const char *fmt,
                                dReal const *a, const int h, const int w,
                                bool pad) {
  printf("%s (%s):\n", name, (pad ? "padded" : "no-pad"));
  for (int row = 0; row < h; row++) {
    for (int col = 0; col < w; col++)
      printf(fmt, a[(pad ? (row * (dPAD(w)) + col) : (row * w + col))]);
    printf("\n");
  }
  printf("\n");
}

#define show_mat(M, X, Y) printMatrixBase(#M, "%f\t", (M),(X),(Y),false)
#define show_Pmat(M, X, Y) printMatrixBase(#M, "%f\t", (M),(X),(Y),true)

__device__ void dQMultiply0 (dQuaternion qa, const dQuaternion qb, const dQuaternion qc) {
  qa[0] = qb[0]*qc[0] - qb[1]*qc[1] - qb[2]*qc[2] - qb[3]*qc[3];
  qa[1] = qb[0]*qc[1] + qb[1]*qc[0] + qb[2]*qc[3] - qb[3]*qc[2];
  qa[2] = qb[0]*qc[2] + qb[2]*qc[0] + qb[3]*qc[1] - qb[1]*qc[3];
  qa[3] = qb[0]*qc[3] + qb[3]*qc[0] + qb[1]*qc[2] - qb[2]*qc[1];
}

__device__ void dDQfromW (dReal dq[4], const dVector3 w, const dQuaternion q)
{
  dq[0] = REAL(0.5)*(- w[0]*q[1] - w[1]*q[2] - w[2]*q[3]);
  dq[1] = REAL(0.5)*(  w[0]*q[0] + w[1]*q[3] - w[2]*q[2]);
  dq[2] = REAL(0.5)*(- w[0]*q[3] + w[1]*q[0] + w[2]*q[1]);
  dq[3] = REAL(0.5)*(  w[0]*q[2] - w[1]*q[1] + w[2]*q[0]);
}

__device__ void dWtoDQ(const dVector3 w, const dQuaternion q, dReal dq[4]) {
	return dDQfromW(dq,w,q);
}

__device__ void dRfromQ (dMatrix3 R, const dQuaternion q)
{
  // q = (s,vx,vy,vz)
  dReal qq1 = 2*q[1]*q[1];
  dReal qq2 = 2*q[2]*q[2];
  dReal qq3 = 2*q[3]*q[3];
  R[(0)*4+(0)] = 1 - qq2 - qq3;
  R[(0)*4+(1)] = 2*(q[1]*q[2] - q[0]*q[3]);
  R[(0)*4+(2)] = 2*(q[1]*q[3] + q[0]*q[2]);
  R[(0)*4+(3)] = (0.0);
  R[(1)*4+(0)] = 2*(q[1]*q[2] + q[0]*q[3]);
  R[(1)*4+(1)] = 1 - qq1 - qq3;
  R[(1)*4+(2)] = 2*(q[2]*q[3] - q[0]*q[1]);
  R[(1)*4+(3)] = (0.0);
  R[(2)*4+(0)] = 2*(q[1]*q[3] - q[0]*q[2]);
  R[(2)*4+(1)] = 2*(q[2]*q[3] + q[0]*q[1]);
  R[(2)*4+(2)] = 1 - qq1 - qq2;
  R[(2)*4+(3)] = (0.0);
}

__device__ void dQtoR(const dQuaternion q, dMatrix3 R) {
	return dRfromQ(R, q);
}

// __device__ dReal dDOTpq(dReal *a, dReal *b, int p, int q) {
// 	return ((a)[0]*(b)[0] + (a)[p]*(b)[q] + (a)[2*(p)]*(b)[2*(q)]);
// }

#define dDOTpq(a,b,p,q)  (a[0]*b[0] + a[p]*b[q] + a[2*(p)]*b[2*(q)])

// __device__ dReal dDOT(dReal *a, dReal *b) {
// 	return dDOTpq(a,b,1,1);
// }


#define dDOT11(a,b) dDOTpq(a,b,1,1)
#define dDOT(a,b) dDOT11(a,b)
#define dDOT13(a,b) dDOTpq(a,b,1,3)
#define dDOT14(a,b) dDOTpq(a,b,1,4)
#define dDOT41(a,b) dDOTpq(a,b,4,1)

__device__ int dNormalize4(dVector4 a) {
  dReal l = dDOT(a,a)+a[3]*a[3];
  if (l > 0) {
    //l = dRecipSqrt(l);
	l = ((1.0f/sqrtf(l)));
    a[0] *= l;
    a[1] *= l;
    a[2] *= l;
    a[3] *= l;
	return 1;
  }
  else {
    a[0] = 1;
    a[1] = 0;
    a[2] = 0;
    a[3] = 0;
    return 0;
  }
}

/*ODE_API void cuda_dxProcessIslands (dxWorld *world, dReal stepsize, dstepper_fn_t cuda_stepper)
{
	int cuda_bodies_count = 0;
	dxBody *cuda_bodies;
	hipMalloc((void**) &cuda_bodies, sizeof(dxBody)*world->nb);
	dxBody *bb;
	for (bb=world->firstbody;bb;bb=(dxBody*)bb->next)
		hipMemcpy(cuda_bodies+sizeof(dxBody)*cuda_bodies_count++, bb, sizeof(dxBody), hipMemcpyHostToDevice);
    cuda_stepper (world,cuda_bodies,cuda_bodies_count,NULL,NULL,stepsize);*/

/* special-case matrix multiplication functions */

// A = B*C  A, B, C all 3x3
// B: pad	A,C: nopad
__device__ void cuda_dMultiply0_333(dReal *A, dReal *B, dReal *C) {

	A[0] = dDOT13((B),(C)); 
	A[1] = dDOT13((B),(C+1)); 
	A[2] = dDOT13((B),(C+2)); 

	A[3] = dDOT13((B+4),(C)); 
	A[4] = dDOT13((B+4),(C+1)); 
	A[5] = dDOT13((B+4),(C+2)); 

	A[6] = dDOT13((B+8),(C)); 
	A[7] = dDOT13((B+8),(C+1)); 
	A[8] = dDOT13((B+8),(C+2)); 
}

// A = B*C^T  A, B, C all 3x3
// A: nopad,	B,C: pad
__device__ void cuda_dMultiply2_333(dReal *A, dReal *B, dReal *C) {
	A[0] = dDOT11((B),(C)); 
	A[1] = dDOT11((B),(C+4)); 
	A[2] = dDOT11((B),(C+8)); 

	A[3] = dDOT11((B+4),(C)); 
	A[4] = dDOT11((B+4),(C+4)); 
	A[5] = dDOT11((B+4),(C+8));

	A[6] = dDOT11((B+8),(C)); 
	A[7] = dDOT11((B+8),(C+4)); 
	A[8] = dDOT11((B+8),(C+8)); 
}

#define CU_ARRAY_DBG(A)                                 \
  printf("---\narray " #A "\n");                        \
  for (i = 0; i < 3; ++i) {                             \
    for (j = 0; j < 3; ++j) {                           \
      printf(#A "[%d,%d]{%f}\t", i, j, (A)[3 * i + j]); \
    }                                                   \
    printf("\n");                                       \
  }

#define MUL_PROC(B, C, OFFSET)                                                 \
  for (j = 0; j < 3; ++j) {                                                    \
    printf("B[%d]*C[%d] = %f * %f = %f\n", j + OFFSET, j, B[j + OFFSET], C[j], \
           B[j + OFFSET] * C[j]);                                              \
  }

// A = B*C  A 3x1, B 3x3, C 3x1
// A,B,C: nopad
__device__ void cuda_dMultiply0_331(dReal *A, dReal  const*B, dReal const*C) {
	A[0] = dDOT11(B, C);
	A[3] = dDOT11((B+3), C);
	A[6] = dDOT11((B+6), C);

	// int i,j;
	// CU_ARRAY_DBG(B);
	// CU_ARRAY_DBG(C);

    // MUL_PROC(B,C,0);
	// MUL_PROC(B,C,3);
	// MUL_PROC(B,C,6);
}

// A = B*C  A 1x3, B 3x3, C 3x1
// A,B,C  nopad
__device__ void cuda_dMultiplyAdd0_331(dReal *A, dReal  const*B, dReal const*C) {
	A[0] += dDOT11(B, C);
	A[1] += dDOT11((B+3), C);
	A[2] += dDOT11((B+6), C);
}

// A = B*C  A 1x3, B 1x3, C 3x3
__device__ void cuda_dMultiply0_133(dReal *A, dReal *B, dReal *C) {
	A[0] = dDOT13((B),(C));
	A[1] = dDOT13((B),(C+1));
	A[2] = dDOT13((B),(C+2));
}


// a -= b cross c
__device__ void cuda_dCross(dReal *a, dReal *b, dReal *c) {
	a[0] -= ((b)[1]*(c)[2] - (b)[2]*(c)[1]);
	a[1] -= ((b)[2]*(c)[0] - (b)[0]*(c)[2]);
	a[2] -= ((b)[0]*(c)[1] - (b)[1]*(c)[0]);
}

// A = B*C  A pxr, B pxq, C qxr
__device__ void naiveMatMultiply(dReal *A, dReal *B, dReal *C, int p, int q, int r) {
  int i, j, k;
  for (j = 0; j < p; ++j)
    for (k = 0; k < r; ++k) {
      for (i = 0; i < q; ++i) {
        A[k + j * r] += B[i] * C[k + r * i];
      }
    }

  // for (i = 0; i < p; i++) {
  // 	for (j = 0; j < r; j++) {
  // 		for (k = 0; k < q; k++) {
  // 			A[i*r + j] += (B[i*q + k])*(C[k*r + j]);
  // 		}
  // 	}
  // }
}

__device__ dReal cuda_sinc(dReal x)
{
	// if |x| < 1e-4 then use a taylor series expansion. this two term expansion
	// is actually accurate to one LS bit within this range if double precision
	// is being used - so don't worry!
	if (fabs(x) < 1.0e-4) return (1.0) - x*x*(0.166666666666666666667);
	else return sinf(x)/x;
}


__device__ void disp_bodyd(dxBody *body) {
  int i;
  printf("flags: %d\n", body->flags);
  printf("mass: %f\n", body->mass);
  printf("InvMass: %f\n", body->invMass);
  printf("posr:\n");
  printf("\tpos: (%f,%f,%f,%f)\n", body->posr.pos[0], body->posr.pos[1],
         body->posr.pos[2], body->posr.pos[3]);
  for (i = 0; i < 3; ++i)
    printf("\tR[%d]: (%f,%f,%f,%f)\n", i, (body->posr.R + i * 4)[0],
           (body->posr.R + i * 4)[1], (body->posr.R + i * 4)[2],
           (body->posr.R + i * 4)[3]);
}

// for debug only
__global__ void cuda_step_none(dxBody *body, int nb, dReal stepsize, dReal g1,
                               dReal g2, dReal g3) {}


// #define _CUDA_DBG
#if defined(_CUDA_DBG)
#define _CUDA_DBG_DO(DO) DO
#else
#define _CUDA_DBG_DO(DO)
#endif


//****************************************************************************
// the slow, but sure way
// note that this does not do any joint feedback!

// given lists of bodies and joints that form an island, perform a first
// order timestep.
//
// `body' is the body array, `nb' is the size of the array.
// `_joint' is the body array, `nj' is the size of the array.
 __global__ void cuda_step(dxBody *body, int nb, dxJoint *joint, int nj, dReal stepsize, dReal g1, dReal g2, dReal g3)
{
	dVector3 gravity; 
	gravity[0] = g1;
	gravity[1] = g2;
	gravity[2] = g3;
	int i,j,k;

	dReal I[3*3], invI[3*3];

	int bid = threadIdx.x + blockDim.x * blockIdx.x;
	if (bid >= nb) { return; }

	// for all bodies, compute the inertia tensor and its inverse in the global
	// frame, and compute the rotational force and add it to the torque
	// accumulator.
	// @@@ check computation of rotational force.

	//dSetZero (I,3*nb*4);
	//dSetZero (invI,3*nb*4);

	_CUDA_DBG_DO(printf("[IN CUDA]================Before: Body[%d]: \n", bid));
	_CUDA_DBG_DO(disp_bodyd(body+bid));

	dReal tmp[9];
#if defined(_CUDA_DBG)
	show_mat(tmp,3,3);
	printf("before compute inertia tensor \n");
	show_Pmat(body[bid].mass.I,3,3);
	show_Pmat(body[bid].posr.R,3,3);
	printf("=== compute inertia tensor \n");
#endif
    // compute inertia tensor in global frame
    cuda_dMultiply2_333(tmp, body[bid].mass.I, body[bid].posr.R);
    cuda_dMultiply0_333(I, body[bid].posr.R, tmp);

	_CUDA_DBG_DO(printf("I after compute inertia tensor \n")); 
	_CUDA_DBG_DO(show_mat(I,3,3);)
	_CUDA_DBG_DO(printf("tmp after compute inertia tensor \n");)
	_CUDA_DBG_DO( show_mat(tmp,3,3);)

if (body[bid].flags & dxBodyGyroscopic) {
    // compute inverse inertia tensor in global frame
    cuda_dMultiply2_333(tmp, body[bid].invI, body[bid].posr.R);
    cuda_dMultiply0_333(invI, body[bid].posr.R, tmp);
#if defined(_CUDA_DBG)
	_CUDA_DBG_DO(printf("tmp after compute inverse inertia tensor \n"); )
	_CUDA_DBG_DO(show_mat(tmp,3,3);)
#endif
	_CUDA_DBG_DO(show_mat(I,3,3));
	_CUDA_DBG_DO(show_mat(body[bid].avel,3,1));
	for(i=0;i<9;++i)tmp[i]=0;

    // compute rotational force
    cuda_dMultiply0_331(tmp, I, body[bid].avel);
	_CUDA_DBG_DO(printf("tmp after compute rotational force \n"));
	_CUDA_DBG_DO(show_mat(tmp,3,3));
    cuda_dCross(body[bid].tacc, body[bid].avel, tmp);
}

	// add the gravity force to all bodies

    if ((body[bid].flags & dxBodyNoGravity)==0) {
		body[bid].facc[0] += body[bid].mass.mass * gravity[0];
		body[bid].facc[1] += body[bid].mass.mass * gravity[1];
		body[bid].facc[2] += body[bid].mass.mass * gravity[2];
    }

	/// Joint relavant implement
	// here
	/// Joint relavant implement

	// create (6*nb,6*nb) inverse mass matrix `invM', and fill it with mass
	// parameters  
	dReal invM[6*6];

	for(i = 0; i < 6*6; i++) invM[i] = 0;



    invM[0] = body[bid].invMass;
    invM[6+1] = body[bid].invMass;
    invM[2*6+2] = body[bid].invMass;
    for (j = 3; j < 6; j++) for (k = 3; k < 6; k++) {
			invM[j*6+k] = invI[j*6+k];
		}
  

	// assemble some body vectors: fe = external forces, v = velocities
	dReal fe[6];
	dReal v[6];

	//dSetZero (fe,n6);
	//dSetZero (v,n6);

	// calculate fe(external force)
    for (j = 0; j < 3; j++) fe[j] = body[bid].facc[j];
    for (j = 0; j < 3; j++) {
		fe[3+j] = body[bid].tacc[j];
		_CUDA_DBG_DO( printf("update body[%d].tacc[%d] = %f\n",bid,j,body[bid].tacc[j]);)
	}

	// this will be set to the velocity update. vnew means cforce
	dReal vnew[6];

	// here should calculate vnew(cforce) according to joints restriction
	// temperarily just set it to zero
	for(i = 0; i < 6; i++) vnew[i] = 0;
	
	// add fe to vnew(cforce)
	for(i = 0; i < 6; i++) vnew[i] += fe[i];
	// // multiply cforce by stepsize
	for(i = 0; i < 6; i++) vnew[i] *= stepsize;



	_CUDA_DBG_DO(show_mat(invM, 6,6);)
	_CUDA_DBG_DO(show_mat(fe, 6,1);)
	_CUDA_DBG_DO(show_mat(vnew, 1, 6));
	// no constraints
	// naiveMatMultiply(vnew, invM, fe, 6, 6, 1);


	// for (i = 0; i < 6; i++) {
	// 	vnew[i] = v[i] + stepsize * vnew[i];
	// 	_CUDA_DBG_DO(printf("update vnew[%d] = %f + %f * %f\n", i, v[i], stepsize, vnew[i]));
	// }

	// apply the velocity update to the bodies

	for (j = 0; j < 3; j++) {
		body[bid].lvel[j] += vnew[j] * (body[bid].invMass);
		_CUDA_DBG_DO( printf("update body[%d].lvel[%d] to %f\n", bid, j, vnew[j]);)
	}
	// add invM * cforce to the body velocity
	cuda_dMultiplyAdd0_331(body[bid].avel, invI, (vnew+3));
    // for (j = 0; j < 3; j++) {
	// 	body[bid].avel[j] += vnew[3 + j];
	// 	_CUDA_DBG_DO(printf("update body[%d].avel[%d] to %f\n", bid, j, vnew[3+j]));
	// }

	// ! impl of void dxStepBody (dxBody *b, dReal h) in CUDA
	// update the position and orientation from the new linear/angular velocity
	// (over the given timestep)
	//dxBody *b = &(body[bid]);

	// cap the angular velocity
	if (body[bid].flags & dxBodyMaxAngularSpeed) {
        const dReal max_ang_speed = body[bid].max_angular_speed;
        const dReal aspeed = dDOT( body[bid].avel, body[bid].avel );
        if (aspeed > max_ang_speed*max_ang_speed) {
			const dReal coef = max_ang_speed/sqrtf(aspeed);
			//dOPEC(b.avel, *=, coef); // multiply vector by scalar coef
			body[bid].avel[0] *= coef;
			body[bid].avel[1] *= coef;
			body[bid].avel[2] *= coef;
        }
	}
	// end of angular velocity cap

	dReal h = stepsize;

	// handle linear velocity
	for (j=0; j<3; j++) body[bid].posr.pos[j] += h * body[bid].lvel[j];

	if (body[bid].flags & dxBodyFlagFiniteRotation) {
		dVector3 irv;	// infitesimal rotation vector
		dQuaternion q;	// quaternion for finite rotation

		if (body[bid].flags & dxBodyFlagFiniteRotationAxis) {
			// split the angular velocity vector into a component along the finite
			// rotation axis, and a component orthogonal to it.
			dVector3 frv;		// finite rotation vector
			dReal k = dDOT (body[bid].finite_rot_axis,body[bid].avel);
			frv[0] = body[bid].finite_rot_axis[0] * k;
			frv[1] = body[bid].finite_rot_axis[1] * k;
			frv[2] = body[bid].finite_rot_axis[2] * k;
			irv[0] = body[bid].avel[0] - frv[0];
			irv[1] = body[bid].avel[1] - frv[1];
			irv[2] = body[bid].avel[2] - frv[2];

			// make a rotation quaternion q that corresponds to frv * h.
			// compare this with the full-finite-rotation case below.
			h *= REAL(0.5);
			dReal theta = k * h;
			q[0] = cosf(theta);
			dReal s = cuda_sinc(theta) * h;
			q[1] = frv[0] * s;
			q[2] = frv[1] * s;
			q[3] = frv[2] * s;
		}
		else {
			// make a rotation quaternion q that corresponds to w * h
			dReal wlen = sqrtf (body[bid].avel[0]*body[bid].avel[0] + body[bid].avel[1]*body[bid].avel[1] +
								body[bid].avel[2]*body[bid].avel[2]);
			h *= REAL(0.5);
			dReal theta = wlen * h;
			q[0] = cosf(theta);
			dReal s = cuda_sinc(theta) * h;
			q[1] = body[bid].avel[0] * s;
			q[2] = body[bid].avel[1] * s;
			q[3] = body[bid].avel[2] * s;
		}

		// do the finite rotation
		dQuaternion q2;
		dQMultiply0 (q2,q,body[bid].q);
		for (j=0; j<4; j++) body[bid].q[j] = q2[j];

		// do the infitesimal rotation if required
		if (body[bid].flags & dxBodyFlagFiniteRotationAxis) {
			dReal dq[4];
			dWtoDQ (irv,body[bid].q,dq);
			for (j=0; j<4; j++) body[bid].q[j] += h * dq[j];
		}
	}
	else {
		// the normal way - do an infitesimal rotation
		dReal dq[4];
		dWtoDQ (body[bid].avel,body[bid].q,dq);
		for (j=0; j<4; j++) body[bid].q[j] += h * dq[j];
	}

	// normalize the quaternion and convert it to a rotation matrix
	dNormalize4 (body[bid].q);
	dQtoR (body[bid].q,body[bid].posr.R);

	// damping
	if (body[bid].flags & dxBodyLinearDamping) {
		const dReal lin_threshold = body[bid].dampingp.linear_threshold;
        const dReal lin_speed = dDOT( body[bid].lvel, body[bid].lvel );
        if ( lin_speed > lin_threshold) {
			const dReal k = 1 - body[bid].dampingp.linear_scale;
			//dOPEC(b.lvel, *=, k);
			body[bid].lvel[0] *= k;
			body[bid].lvel[1] *= k;
			body[bid].lvel[2] *= k;
        }
	}
	if (body[bid].flags & dxBodyAngularDamping) {
		const dReal ang_threshold = body[bid].dampingp.angular_threshold;
        const dReal ang_speed = dDOT( body[bid].avel, body[bid].avel );
        if ( ang_speed > ang_threshold) {
			const dReal k = 1 - body[bid].dampingp.angular_scale;
			//dOPEC(b.avel, *=, k);
			body[bid].avel[0] *= k;
			body[bid].avel[1] *= k;
			body[bid].avel[2] *= k;
        }
	}

	// zero all force accumulators
    body[bid].facc[0] = 0;
    body[bid].facc[1] = 0;
    body[bid].facc[2] = 0;
    body[bid].facc[3] = 0;
    body[bid].tacc[0] = 0;
    body[bid].tacc[1] = 0;
    body[bid].tacc[2] = 0;
    body[bid].tacc[3] = 0;

	_CUDA_DBG_DO( printf("================After: Body[%d]: \n", bid);)
	_CUDA_DBG_DO(disp_bodyd(body+bid);)

}

ODE_API void cuda_dInternalStepIsland_x1 (dxWorld *world, dxBody *cuda_body, int nb, dxJoint *_joint, int nj, dReal stepsize)
{

	cuda_step<<<nb, 1>>>(cuda_body, world->nb, _joint, world->nj, stepsize, world->gravity[0], world->gravity[1], world->gravity[2]);
	// cuda_step<<<1, 1>>>(cuda_body, world->nb, stepsize, world->gravity[0], world->gravity[1], world->gravity[2]);

	//cuda_step<<<BLOCKSIZE/nb, 256>>>(cuda_body, world->nb, stepsize, world->gravity[0], world->gravity[1], world->gravity[2]);
}

 ODE_API void cuda_dxProcessIslands(dxWorld *world, dxBody *cuda_body, dxJoint *cuda_joint, dReal stepsize, dstepper_fn_t stepper)
{
	const int block_size = BLOCKSIZE;
	dim3 dimBlock(block_size, block_size);
	dim3 dimGrid(block_size, block_size);

	cuda_dInternalStepIsland_x1(world, cuda_body, world->nb, cuda_joint, world->nj, stepsize);
}

