#include "hip/hip_runtime.h"
// cuda_step.cu

#include "objects.h"
#include "joints/joint.h"
#include <ode/odeconfig.h>
#include "config.h"
#include <ode/odemath.h>
#include <ode/rotation.h>
#include <ode/timer.h>
#include <ode/error.h>
#include <ode/matrix.h>
#include "lcp.h"
#include "util.h"

#include <ode/cuda_step.h>
#include <hip/hip_runtime.h>
#include <ode/cuda_helper.h>
#include <ode/cuda_matrix.h>

/*ODE_API void cuda_dxProcessIslands (dxWorld *world, dReal stepsize, dstepper_fn_t cuda_stepper)
{
	int cuda_bodies_count = 0;
	dxBody *cuda_bodies;
	hipMalloc((void**) &cuda_bodies, sizeof(dxBody)*world->nb);
	dxBody *bb;
	for (bb=world->firstbody;bb;bb=(dxBody*)bb->next)
		hipMemcpy(cuda_bodies+sizeof(dxBody)*cuda_bodies_count++, bb, sizeof(dxBody), hipMemcpyHostToDevice);
    cuda_stepper (world,cuda_bodies,cuda_bodies_count,NULL,NULL,stepsize);*/

/* special-case matrix multiplication functions */

// A = B*C  A, B, C all 3x3
__device__ void cuda_dMultiply0_333(dReal *A, dReal *B, dReal *C) {
	A[0] = dDOT13((B),(C)); 
	A[1] = dDOT13((B),(C+1)); 
	A[2] = dDOT13((B),(C+2)); 
	A[4] = dDOT13((B+4),(C)); 
	A[5] = dDOT13((B+4),(C+1)); 
	A[6] = dDOT13((B+4),(C+2));
	A[8] = dDOT13((B+8),(C)); 
	A[9] = dDOT13((B+8),(C+1)); 
	A[10] = dDOT13((B+8),(C+2)); 
}

// A = B*C^T  A, B, C all 3x3
__device__ void cuda_dMultiply2_333(dReal *A, dReal *B, dReal *C) {
	A[0] = dDOT((B),(C)); 
	A[1] = dDOT((B),(C+4)); 
	A[2] = dDOT((B),(C+8)); 
	A[4] = dDOT((B+4),(C)); 
	A[5] = dDOT((B+4),(C+4)); 
	A[6] = dDOT((B+4),(C+8));
	A[8] = dDOT((B+8),(C)); 
	A[9] = dDOT((B+8),(C+4)); 
	A[10] = dDOT((B+8),(C+8)); 
}

// A = B*C  A 3x1, B 3x3, C 3x1
__device__ void cuda_dMultiply0_331(dReal *A, dReal *B, dReal *C) {
	A[0] = dDOT((B),(C));
	A[1] = dDOT((B+4),(C));
	A[2] = dDOT((B+8),(C));
}

// A = B*C  A 1x3, B 1x3, C 3x3
__device__ void cuda_dMultiply0_133(dReal *A, dReal *B, dReal *C) {
	A[0] = dDOT13((B),(C));
	A[1] = dDOT13((B),(C+1));
	A[2] = dDOT13((B),(C+2));
}

// A += B*C  A 3x1, B 3x3, C 3x1
__device__ void cuda_dMultiplyAdd0_331(dReal *A, dReal *B, dReal *C) {
	A[0] += dDOT((B),(C));
	A[1] += dDOT((B+4),(C));
	A[2] += dDOT((B+8),(C));
}

// a -= b cross c
__device__ void cuda_dCross(dReal *a, dReal *b, dReal *c) {
	a[0] -= ((b)[1]*(c)[2] - (b)[2]*(c)[1]);
	a[1] -= ((b)[2]*(c)[0] - (b)[0]*(c)[2]);
	a[2] -= ((b)[0]*(c)[1] - (b)[1]*(c)[0]);
}

// A = B*C  A pxr, B pxq, C qxr
__device__ void naiveMatMultiply(dReal *A, dReal *B, dReal *C, int p, int q, int r) {
	int i, j, k;
	for (i = 0; i < p; i++) {
		for (j = 0; j < r; j++) {
			for (k = 0; k < q; k++) {
				A[i*r + j] += (B[i*q + k])*(C[k*r + j]);
			}
		}
	}
}

-__device__ dReal cuda_sinc(dReal x)
{
	// if |x| < 1e-4 then use a taylor series expansion. this two term expansion
	// is actually accurate to one LS bit within this range if double precision
	// is being used - so don't worry!
	if (dFabs(x) < 1.0e-4) return REAL(1.0) - x*x*REAL(0.166666666666666666667);
	else return dSin(x)/x;
}

//****************************************************************************
// the slow, but sure way
// note that this does not do any joint feedback!

// given lists of bodies and joints that form an island, perform a first
// order timestep.
//
// `body' is the body array, `nb' is the size of the array.
// `_joint' is the body array, `nj' is the size of the array.

__global__ void cuda_step(dxWorld *world, dxBody * const *body, int nb, dxJoint * const *_joint, int nj, dReal stepsize)
{
	int i,j,k;

	dReal I[3][3], invI[3][3];

	int bid = blockIdx.x;

	// for all bodies, compute the inertia tensor and its inverse in the global
	// frame, and compute the rotational force and add it to the torque
	// accumulator.
	// @@@ check computation of rotational force.

	//dSetZero (I,3*nb*4);
	//dSetZero (invI,3*nb*4);
	dReal tmp[9];


    // compute inertia tensor in global frame
    cuda_dMultiply2_333(tmp, body[bid]->mass.I, body[bid]->posr.R);
    cuda_dMultiply0_333(I, body[bid]->posr.R, tmp);
    // compute inverse inertia tensor in global frame
    cuda_dMultiply2_333(tmp, body[bid]->invI, body[bid]->posr.R);
    cuda_dMultiply0_333(invI, body[bid]->posr.R, tmp);
    // compute rotational force
    cuda_dMultiply0_331(tmp, I, body[bid]->avel);
    cuda_dCross(body[bid]->tacc, body[bid]->avel, tmp);


	// add the gravity force to all bodies

    if ((body[bid]->flags & dxBodyNoGravity)==0) {
		body[bid]->facc[0] += body[bid]->mass.mass * world->gravity[0];
		body[bid]->facc[1] += body[bid]->mass.mass * world->gravity[1];
		body[bid]->facc[2] += body[bid]->mass.mass * world->gravity[2];
    }


	// create (6*nb,6*nb) inverse mass matrix `invM', and fill it with mass
	// parameters  
	dReal invM[6][6];

	for(i = 0; i < 6*6; i++) invM[i] = 0;



    invM[0] = body[bid]->invMass;
    invM[6+1] = body[bid]->invMass;
    invM[2*6+2] = body[bid]->invMass;
    for (j = 3; j < 6; j++) for (k = 3; k < 6; k++) {
			invM[j*6+k] = invI[j*6+k];
		}
  

	// assemble some body vectors: fe = external forces, v = velocities
	dReal fe[6];
	dReal v[6];

	//dSetZero (fe,n6);
	//dSetZero (v,n6);

    for (j = 0; j < 3; j++) fe[j] = body[bid]->facc[j];
    for (j = 0; j < 3; j++) fe[3+j] = body[bid]->tacc[j];
    for (j = 0; j < 3; j++) v[j] = body[bid]->lvel[j];
    for (j = 0; j < 3; j++) v[3+j] = body[bid]->avel[j];

	// this will be set to the velocity update
	dReal vnew[6];
	for(i = 0; i < 6; i++) vnew[i] = 0;

	// no constraints
	naiveMatMultiply(vnew, invM, fe, 6, 6, 1);
	for (i = 0; i < 6; i++) vnew[i] = v[i] + stepsize*vnew[i];

	// apply the velocity update to the bodies

    for (j = 0; j < 3; j++) body[bid]->lvel[j] = vnew[j];
    for (j = 0; j < 3; j++) body[bid]->avel[j] = vnew[3+j];


	// update the position and orientation from the new linear/angular velocity
	// (over the given timestep)
	dxBody *b = &(body[bid]);
	// cap the angular velocity
	if (b->flags & dxBodyMaxAngularSpeed) {
        const dReal max_ang_speed = b->max_angular_speed;
        const dReal aspeed = dDOT( b->avel, b->avel );
        if (aspeed > max_ang_speed*max_ang_speed) {
			const dReal coef = max_ang_speed/dSqrt(aspeed);
			dOPEC(b->avel, *=, coef); // multiply vector by scalar coef
        }
	}
	// end of angular velocity cap


	// handle linear velocity
	for (j=0; j<3; j++) b->posr.pos[j] += h * b->lvel[j];

	if (b->flags & dxBodyFlagFiniteRotation) {
		dVector3 irv;	// infitesimal rotation vector
		dQuaternion q;	// quaternion for finite rotation

		if (b->flags & dxBodyFlagFiniteRotationAxis) {
			// split the angular velocity vector into a component along the finite
			// rotation axis, and a component orthogonal to it.
			dVector3 frv;		// finite rotation vector
			dReal k = dDOT (b->finite_rot_axis,b->avel);
			frv[0] = b->finite_rot_axis[0] * k;
			frv[1] = b->finite_rot_axis[1] * k;
			frv[2] = b->finite_rot_axis[2] * k;
			irv[0] = b->avel[0] - frv[0];
			irv[1] = b->avel[1] - frv[1];
			irv[2] = b->avel[2] - frv[2];

			// make a rotation quaternion q that corresponds to frv * h.
			// compare this with the full-finite-rotation case below.
			h *= REAL(0.5);
			*      dReal theta = k * h;
			q[0] = dCos(theta);
			dReal s = cuda_sinc(theta) * h;
			q[1] = frv[0] * s;
			q[2] = frv[1] * s;
			q[3] = frv[2] * s;
		}
		else {
			// make a rotation quaternion q that corresponds to w * h
			dReal wlen = dSqrt (b->avel[0]*b->avel[0] + b->avel[1]*b->avel[1] +
								b->avel[2]*b->avel[2]);
			h *= REAL(0.5);
			dReal theta = wlen * h;
			q[0] = dCos(theta);
			dReal s = cuda_sinc(theta) * h;
			q[1] = b->avel[0] * s;
			q[2] = b->avel[1] * s;
			q[3] = b->avel[2] * s;
		}

		// do the finite rotation
		dQuaternion q2;
		dQMultiply0 (q2,q,b->q);
		for (j=0; j<4; j++) b->q[j] = q2[j];

		// do the infitesimal rotation if required
		if (b->flags & dxBodyFlagFiniteRotationAxis) {
			dReal dq[4];
			dWtoDQ (irv,b->q,dq);
			for (j=0; j<4; j++) b->q[j] += h * dq[j];
		}
	}
	else {
		// the normal way - do an infitesimal rotation
		dReal dq[4];
		dWtoDQ (b->avel,b->q,dq);
		for (j=0; j<4; j++) b->q[j] += h * dq[j];
	}

	// normalize the quaternion and convert it to a rotation matrix
	dNormalize4 (b->q);
	dQtoR (b->q,b->posr.R);

	// damping
	if (b->flags & dxBodyLinearDamping) {
		const dReal lin_threshold = b->dampingp.linear_threshold;
        const dReal lin_speed = dDOT( b->lvel, b->lvel );
        if ( lin_speed > lin_threshold) {
			const dReal k = 1 - b->dampingp.linear_scale;
			dOPEC(b->lvel, *=, k);
        }
	}
	if (b->flags & dxBodyAngularDamping) {
		const dReal ang_threshold = b->dampingp.angular_threshold;
        const dReal ang_speed = dDOT( b->avel, b->avel );
        if ( ang_speed > ang_threshold) {
			const dReal k = 1 - b->dampingp.angular_scale;
			dOPEC(b->avel, *=, k);
        }
	}


	// zero all force accumulators
    body[bid]->facc[0] = 0;
    body[bid]->facc[1] = 0;
    body[bid]->facc[2] = 0;
    body[bid]->facc[3] = 0;
    body[bid]->tacc[0] = 0;
    body[bid]->tacc[1] = 0;
    body[bid]->tacc[2] = 0;
    body[bid]->tacc[3] = 0;
}
