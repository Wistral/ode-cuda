#include "hip/hip_runtime.h"
// cuda_step.cu

#include "objects.h"
#include "joints/joint.h"
#include <ode/odeconfig.h>
#include "config.h"
#include <ode/odemath.h>
#include <ode/rotation.h>
#include <ode/timer.h>
#include <ode/error.h>
#include <ode/matrix.h>
#include "lcp.h"
#include "util.h"

#include <ode/cuda_step.h>
#include <hip/hip_runtime.h>
#include <ode/cuda_helper.h>
#include <ode/cuda_matrix.h>



#define ALLOCA(t,v,s)                           \
  Auto<t> v( dALLOCA16(s) );


//****************************************************************************
// the slow, but sure way
// note that this does not do any joint feedback!

// given lists of bodies and joints that form an island, perform a first
// order timestep.
//
// `body' is the body array, `nb' is the size of the array.
// `_joint' is the body array, `nj' is the size of the array.

__global__ void cuda_step(dxWorld *world, dxBody * const *body, int nb, dReal stepsize)
{
  int i,j,k;
  int n6 = 6*nb;

  // for all bodies, compute the inertia tensor and its inverse in the global
  // frame, and compute the rotational force and add it to the torque
  // accumulator.
  // @@@ check computation of rotational force.
  ALLOCA(dReal,I,3*nb*4*sizeof(dReal));
  ALLOCA(dReal,invI,3*nb*4*sizeof(dReal));

  //dSetZero (I,3*nb*4);
  //dSetZero (invI,3*nb*4);
  for (i=0; i<nb; i++) {
    dReal tmp[12];
    // compute inertia tensor in global frame
    dMULTIPLY2_333 (tmp,body[i]->mass.I,body[i]->posr.R);
    dMULTIPLY0_333 (I+i*12,body[i]->posr.R,tmp);
    // compute inverse inertia tensor in global frame
    dMULTIPLY2_333 (tmp,body[i]->invI,body[i]->posr.R);
    dMULTIPLY0_333 (invI+i*12,body[i]->posr.R,tmp);
    // compute rotational force
    dMULTIPLY0_331 (tmp,I+i*12,body[i]->avel);
    dCROSS (body[i]->tacc,-=,body[i]->avel,tmp);
  }

  // add the gravity force to all bodies
  for (i=0; i<nb; i++) {
    if ((body[i]->flags & dxBodyNoGravity)==0) {
      body[i]->facc[0] += body[i]->mass.mass * world->gravity[0];
      body[i]->facc[1] += body[i]->mass.mass * world->gravity[1];
      body[i]->facc[2] += body[i]->mass.mass * world->gravity[2];
    }
  }
  // create (6*nb,6*nb) inverse mass matrix `invM', and fill it with mass
  // parameters

  int nskip = dPAD (n6);
  ALLOCA(dReal, invM, n6*nskip*sizeof(dReal));
  
  dSetZero (invM,n6*nskip);
  for (i=0; i<nb; i++) {
    dReal *MM = invM+(i*6)*nskip+(i*6);
    MM[0] = body[i]->invMass;
    MM[nskip+1] = body[i]->invMass;
    MM[2*nskip+2] = body[i]->invMass;
    MM += 3*nskip+3;
    for (j=0; j<3; j++) for (k=0; k<3; k++) {
      MM[j*nskip+k] = invI[i*12+j*4+k];
    }
  }

  // assemble some body vectors: fe = external forces, v = velocities
  ALLOCA(dReal,fe,n6*sizeof(dReal));
  ALLOCA(dReal,v,n6*sizeof(dReal));

  //dSetZero (fe,n6);
  //dSetZero (v,n6);
  for (i=0; i<nb; i++) {
    for (j=0; j<3; j++) fe[i*6+j] = body[i]->facc[j];
    for (j=0; j<3; j++) fe[i*6+3+j] = body[i]->tacc[j];
    for (j=0; j<3; j++) v[i*6+j] = body[i]->lvel[j];
    for (j=0; j<3; j++) v[i*6+3+j] = body[i]->avel[j];
  }

  // this will be set to the velocity update
  ALLOCA(dReal,vnew,n6*sizeof(dReal));
  dSetZero (vnew,n6);

  // no constraints
  dMultiply0 (vnew,invM,fe,n6,n6,1);
  for (i=0; i<n6; i++) vnew[i] = v[i] + stepsize*vnew[i];

  // apply the velocity update to the bodies
  for (i=0; i<nb; i++) {
    for (j=0; j<3; j++) body[i]->lvel[j] = vnew[i*6+j];
    for (j=0; j<3; j++) body[i]->avel[j] = vnew[i*6+3+j];
  }

  // update the position and orientation from the new linear/angular velocity
  // (over the given timestep)
  for (i=0; i<nb; i++) dxStepBody (body[i],stepsize);

  // zero all force accumulators
  for (i=0; i<nb; i++) {
    body[i]->facc[0] = 0;
    body[i]->facc[1] = 0;
    body[i]->facc[2] = 0;
    body[i]->facc[3] = 0;
    body[i]->tacc[0] = 0;
    body[i]->tacc[1] = 0;
    body[i]->tacc[2] = 0;
    body[i]->tacc[3] = 0;
  }
}
